#include "hip/hip_runtime.h"
﻿
// taken from OptiX_Utility
// https://github.com/shocker-0x15/OptiX_Utility/blob/master/LICENSE.md
// and from Shocker GfxExp
// https://github.com/shocker-0x15/GfxExp

#include "../Shared.h"
#include <common_device.h>

using namespace Shared;
using namespace shared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters plp;

class LambertBRDF
{
    RGB m_reflectance; // The diffuse reflectance of the surface

 public:
    // Constructor: Initializes the Lambertian BRDF with a given reflectance value
    CUDA_DEVICE_FUNCTION LambertBRDF (const RGB& reflectance) :
        m_reflectance (reflectance) {}

    // Get surface parameters like diffuse and specular reflectance, and roughness
    CUDA_DEVICE_FUNCTION void getSurfaceParameters (RGB* diffuseReflectance, RGB* specularReflectance, float* roughness) const
    {
        *diffuseReflectance = m_reflectance;           // Lambertian surfaces are purely diffuse
        *specularReflectance = RGB (0.0f, 0.0f, 0.0f); // No specular reflection
        *roughness = 1.0f;                             // Lambertian surfaces are completely rough
    }

    // Sample throughput for a given incoming direction
    // Also samples an outgoing direction based on input random variables uDir0 and uDir1
    CUDA_DEVICE_FUNCTION RGB sampleThroughput (
        const Vector3D& vGiven, float uDir0, float uDir1,
        Vector3D* vSampled, float* dirPDensity) const
    {
        *vSampled = cosineSampleHemisphere (uDir0, uDir1); // Sample direction in hemisphere
        *dirPDensity = vSampled->z / Pi;                   // Probability density for the sampled direction
        if (vGiven.z <= 0.0f)                              // Flip z if incoming direction is below the surface
            vSampled->z *= -1;
        return m_reflectance; // Return the reflectance value
    }

    // Evaluate the BRDF for a given incoming and outgoing direction
    CUDA_DEVICE_FUNCTION RGB evaluate (const Vector3D& vGiven, const Vector3D& vSampled) const
    {
        if (vGiven.z * vSampled.z > 0) // Both vectors should be on the same side of the surface
            return m_reflectance / Pi;
        else
            return RGB (0.0f, 0.0f, 0.0f); // Zero if vectors are on opposite sides
    }

    // Evaluate the PDF for the BRDF for given directions
    CUDA_DEVICE_FUNCTION float evaluatePDF (const Vector3D& vGiven, const Vector3D& vSampled) const
    {
        if (vGiven.z * vSampled.z > 0)
            return fabs (vSampled.z) / Pi;
        else
            return 0.0f; // Zero PDF if vectors are on opposite sides
    }

    // Estimate the directional-hemispherical reflectance for a given direction
    CUDA_DEVICE_FUNCTION RGB evaluateDHReflectanceEstimate (const Vector3D& vGiven) const
    {
        return m_reflectance; // For Lambertian, it's just the reflectance
    }
};

// This function is for computing the direct lighting on a surface point (shadingPoint)
// from an environment light source (lightSample) using a Lambertian BRDF (bsdf).
// It takes into account visibility, distances, and angles to compute the final light
// contribution at that point.
CUDA_DEVICE_FUNCTION CUDA_INLINE RGB computeDirectLightingFromEnvironment (
    const Point3D& shadingPoint, const Vector3D& vOutLocal, const ReferenceFrame& shadingFrame,
    const LambertBRDF& bsdf, const Shared::LightSample& lightSample)
{
    // Calculate the direction of the shadow ray
    Vector3D shadowRayDir = lightSample.atInfinity ? Vector3D (lightSample.position) : (lightSample.position - shadingPoint);

    // Calculate the distance squared and distance between the light and the shading point
    float dist2 = shadowRayDir.sqLength();
    float dist = std::sqrt (dist2);

    // Normalize the shadow ray direction
    shadowRayDir /= dist;

    // Convert shadow ray direction to local coordinate system
    Vector3D shadowRayDirLocal = shadingFrame.toLocal (shadowRayDir);

    // Compute the cosine of the angle between the light direction and light normal
    float lpCos = dot (-shadowRayDir, lightSample.normal);

    // Compute the cosine of the angle between shadow ray and normal at the shading point in local coords
    float spCos = shadowRayDirLocal.z;

    // Initialize visibility to 1 (completely visible)
    float visibility = 1.0f;

    // Set a high distance for lights at infinity
    if (lightSample.atInfinity)
        dist = 1e+10f;

    // Perform visibility ray tracing to check if the light is occluded
    Shared::VisibilityRayPayloadSignature::trace (
        plp.travHandle,
        shadingPoint.toNative(), shadowRayDir.toNative(), 0.0f, dist * 0.9999f, 0.0f,
        0xFF, OPTIX_RAY_FLAG_NONE,
        RayType::RayType_Visibility, Shared::NumRayTypes, RayType::RayType_Visibility,
        visibility);

    // If the point is visible and faces the light
    if (visibility > 0 && lpCos > 0)
    {
        // Calculate emittance assuming the light is a diffuse emitter
        RGB Le = lightSample.emittance / Pi;

        // Evaluate the Lambertian BRDF
        RGB fsValue = bsdf.evaluate (vOutLocal, shadowRayDirLocal);

        // Calculate the geometry term
        float G = lpCos * std::fabs (spCos) / dist2;

        // Final lighting contribution
        RGB ret = fsValue * Le * G;
        return ret;
    }
    else
    {
        // Return black if the point is not visible or does not face the light
        return RGB (0.0f, 0.0f, 0.0f);
    }
}

// This function samples an environmental light based on a set of
// random numbers (u0 and u1) and an importance map.It returns the
// sampled light direction, emittance, and some other attributes
// in lightSample.It also returns the probability density of the
// sampled area in areaPDensity.
CUDA_DEVICE_FUNCTION CUDA_INLINE void sampleEnviroLight (
    const Point3D& shadingPoint,
    float ul, bool sampleEnvLight, float u0, float u1,
    Shared::LightSample* lightSample, float* areaPDensity)
{
    hipTextureObject_t texEmittance = 0;     // Texture object for light emittance
    RGB emittance (0.0f, 0.0f, 0.0f); // Light emittance color
    Point2D texCoord;                 // Texture coordinates

    float u, v;  // Parameters for sampling
    float uvPDF; // PDF for UV sampling

    // Sample the importance map to get UV coordinates and PDF
    plp.envLightImportanceMap.sample (u0, u1, &u, &v, &uvPDF);

    // Convert UV to spherical coordinates
    float phi = 2 * Pi * u;
    float theta = Pi * v;
    if (theta == 0.0f)
    {
        *areaPDensity = 0.0f;
        return;
    }

    // Apply rotation to the environment light
    float posPhi = phi - plp.envLightRotation;
    posPhi = posPhi - floorf (posPhi / (2 * Pi)) * 2 * Pi;

    // Convert spherical to Cartesian coordinates
    Vector3D direction = fromPolarYUp (posPhi, theta);
    Point3D position (direction.x, direction.y, direction.z);

    // Set light sample attributes
    lightSample->position = position;
    lightSample->atInfinity = true;
    lightSample->normal = Normal3D (-position);

    // Compute the area PDF
    *areaPDensity = uvPDF / (2 * Pi * Pi * std::sin (theta));

    // Retrieve the environment light texture
    texEmittance = plp.envLightTexture;

    // Set a base emittance value
    emittance = RGB (Pi * plp.envLightPowerCoeff);
    texCoord.x = u;
    texCoord.y = v;

    // If a texture is available, update emittance based on texture values
    if (texEmittance)
    {
        float4 texValue = tex2DLod<float4> (texEmittance, texCoord.x, texCoord.y, 0.0f);
        emittance *= RGB (texValue.x, texValue.y, texValue.z);
    }

    // Set the emittance in the light sample
    lightSample->emittance = emittance;
}

// Next Event Estimation (NEE) is a technique used in path tracing to improve
// the convergence of the rendered image. Instead of randomly bouncing rays around the scene,
// NEE takes a shortcut and directly samples a light source to check if it contributes to
// the illumination of a point.

// In a traditional path tracer, rays are shot from the camera and bounce around the scene
// until they hit a light source. This can take many bounces and lead to a noisy image.

// With NEE, when a ray hits a surface, the algorithm also sends a direct ray to a light source
// to see if it's visible from that point. This helps to quickly account for direct illumination,
// making the image converge faster and reducing noise.

// This function is for performing Next Event Estimation (NEE) in path tracing.
// It samples a light source, computes the direct lighting from that source,
// and combines it with the BRDF and visibility information.The function also
// uses Multiple Importance Sampling (MIS)
// to balance the contributions from the BRDF and the light source.
CUDA_DEVICE_FUNCTION CUDA_INLINE RGB performNextEventEstimation (
    const Point3D& shadingPoint, const Vector3D& vOutLocal, const ReferenceFrame& shadingFrame,
    const LambertBRDF& bsdf,
    PCG32RNG& rng)
{
    RGB ret (0.0f); // Initialize the return value

    // Generate a random number to select a light source
    float uLight = rng.getFloat0cTo1o();
    bool selectEnvLight = true;
    float probToSampleCurLightType = 1.0f; // Probability of sampling the current light type

    LightSample lightSample; // Sampled light information
    float areaPDensity;      // Area probability density
    // Sample the environmental light source
    sampleEnviroLight (
        shadingPoint,
        uLight, selectEnvLight, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &lightSample, &areaPDensity);

    areaPDensity *= probToSampleCurLightType; // Update the area PDF with the light type selection probability

    float misWeight = 1.0f; // Multiple Importance Sampling (MIS) weight

    // Calculate the shadow ray direction
    Vector3D shadowRay = lightSample.atInfinity ? Vector3D (lightSample.position) : (lightSample.position - shadingPoint);
    float dist2 = shadowRay.sqLength();                   // Distance squared to the light
    shadowRay /= std::sqrt (dist2);                       // Normalize the shadow ray
    Vector3D vInLocal = shadingFrame.toLocal (shadowRay); // Convert to local coordinates

    // Calculate the cosine term and BSDF PDF
    float lpCos = std::fabs (dot (shadowRay, lightSample.normal));
    float bsdfPDensity = bsdf.evaluatePDF (vOutLocal, vInLocal) * lpCos / dist2; // BSDF PDF
    if (!isfinite (bsdfPDensity))                                                // Check for invalid values
        bsdfPDensity = 0.0f;

    // Calculate the light source PDF and MIS weight
    float lightPDensity = areaPDensity;
    misWeight = pow2 (lightPDensity) / (pow2 (bsdfPDensity) + pow2 (lightPDensity));

    // Compute the direct lighting contribution if the area PDF is positive
    if (areaPDensity > 0.0f)
        ret = computeDirectLightingFromEnvironment (
                  shadingPoint, vOutLocal, shadingFrame, bsdf, lightSample) *
              (misWeight / areaPDensity);

    return ret; // Return the final lighting contribution
}

// This function calculates various attributes of a surface point
// given its barycentric coordinates (b1, b2) and the index (primIndex)
// of the triangle it belongs to. It computes the world-space position,
// shading normal, texture coordinates, and so forth for this surface point.
// It also computes a hypothetical area PDF (hypAreaPDensity) that could
// be used in light sampling.
CUDA_DEVICE_FUNCTION CUDA_INLINE void computeSurfacePoint (
    const Shared::GeometryData& geomInst,
    uint32_t primIndex, float b1, float b2,
    const Point3D& referencePoint,
    Point3D* positionInWorld, Normal3D* shadingNormalInWorld, Vector3D* texCoord0DirInWorld,
    Normal3D* geometricNormalInWorld, Point2D* texCoord,
    float* hypAreaPDensity)
{
    // Fetch the vertices of the triangle given its index
    const Triangle& tri = geomInst.triangleBuffer[primIndex];
    const Vertex& v0 = geomInst.vertexBuffer[tri.index0];
    const Vertex& v1 = geomInst.vertexBuffer[tri.index1];
    const Vertex& v2 = geomInst.vertexBuffer[tri.index2];

    // Transform vertex positions to world space
    const Point3D p[3] = {
        transformPointFromObjectToWorldSpace (v0.position),
        transformPointFromObjectToWorldSpace (v1.position),
        transformPointFromObjectToWorldSpace (v2.position),
    };

    // Calculate barycentric coordinates
    float b0 = 1 - (b1 + b2);

    // Compute the position in world space using barycentric coordinates
    *positionInWorld = b0 * p[0] + b1 * p[1] + b2 * p[2];

    // Compute interpolated shading normal and texture direction
    Normal3D shadingNormal = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
    Vector3D texCoord0Dir = b0 * v0.texCoord0Dir + b1 * v1.texCoord0Dir + b2 * v2.texCoord0Dir;

    // Compute geometric normal and area of the triangle
    Normal3D geometricNormal (cross (p[1] - p[0], p[2] - p[0]));
    float area = 0.5f * length (geometricNormal);

    // Compute the texture coordinates
    *texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

    // Transform shading normal and texture direction to world space
    *shadingNormalInWorld = normalize (transformNormalFromObjectToWorldSpace (shadingNormal));
    *texCoord0DirInWorld = normalize (transformVectorFromObjectToWorldSpace (texCoord0Dir));
    *geometricNormalInWorld = normalize (geometricNormal);

    // Check for invalid normals and correct them
    if (!shadingNormalInWorld->allFinite())
    {
        *shadingNormalInWorld = Normal3D (0, 0, 1);
        *texCoord0DirInWorld = Vector3D (1, 0, 0);
    }

    // Check for invalid texture directions and correct them
    if (!texCoord0DirInWorld->allFinite())
    {
        Vector3D bitangent;
        makeCoordinateSystem (*shadingNormalInWorld, texCoord0DirInWorld, &bitangent);
    }

    // Compute the probability of sampling this light
    float lightProb = 1.0f;
    if (plp.envLightTexture && plp.enableEnvLight)
        lightProb *= (1 - probToSampleEnvLight);

    // Check for invalid probabilities
    if (!isfinite (lightProb))
    {
        *hypAreaPDensity = 0.0f;
        return;
    }

    // Compute the hypothetical area PDF
    *hypAreaPDensity = lightProb / area;
}

// Define a struct called HitPointParameter to hold hit point info
struct HitPointParameter
{
    float b1, b2;      // Barycentric coordinates
    int32_t primIndex; // Index of the primitive hit by the ray

    // Static member function to get hit point parameters
    CUDA_DEVICE_FUNCTION CUDA_INLINE static HitPointParameter get()
    {
        HitPointParameter ret; // Create an instance of the struct

        // Get barycentric coordinates from OptiX API
        float2 bc = optixGetTriangleBarycentrics();

        // Store the barycentric coordinates in the struct
        ret.b1 = bc.x;
        ret.b2 = bc.y;

        // Get the index of the primitive hit by the ray from OptiX API
        ret.primIndex = optixGetPrimitiveIndex();

        // Return the populated struct
        return ret;
    }
};

// This struct is used to fetch geometry and material data from
// the Shader Binding Table (SBT) in OptiX.
struct HitGroupSBTRecordData
{
    GeometryData geomData;        // Geometry data for the hit object
    Shared::MaterialData matData; // Material data for the hit object

    // Static member function to retrieve the SBT record data
    CUDA_DEVICE_FUNCTION CUDA_INLINE static const HitGroupSBTRecordData& get()
    {
        // Use optixGetSbtDataPointer() to get the pointer to the SBT data
        // Cast the pointer to type HitGroupSBTRecordData and dereference it
        return *reinterpret_cast<HitGroupSBTRecordData*> (optixGetSbtDataPointer());
    }
};

// Define the ray generating kernel for path tracing
CUDA_DEVICE_KERNEL void RT_RG_NAME (pathTracing)()
{
    // Get the launch index for this thread
    uint2 launchIndex = make_uint2 (optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    // Initialize the random number generator
    PCG32RNG rng = plp.rngBuffer[launchIndex];

    // Get camera properties from the pipeline parameters
    const PerspectiveCamera& camera = plp.camera;

    // Generate random numbers for jittering
    float jx = rng.getFloat0cTo1o();
    float jy = rng.getFloat0cTo1o();

    // Update the RNG buffer
    plp.rngBuffer.write (launchIndex, rng);

    // Compute normalized screen coordinates
    float x = (launchIndex.x + jx) / plp.imageSize.x;
    float y = (launchIndex.y + jy) / plp.imageSize.y;

    // Compute vertical and horizontal view angles
    float vh = 2 * std::tan (plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    // Setup ray origin and direction
    Point3D origin = camera.position;
    Vector3D direction = normalize (camera.orientation * Vector3D (vw * (0.5f - x), vh * (0.5f - y), 1));

    // Initialize ray payload
    SearchRayPayload payload;
    payload.alpha = RGB (1.0f, 1.0f, 1.0f);
    payload.contribution = RGB (0.0f, 0.0f, 0.0f);
    payload.pathLength = 1;
    payload.terminate = false;
    SearchRayPayload* payloadPtr = &payload;

    RGB firstHitAlbedo (0.0f, 0.0f, 0.0f);
    Normal3D firstHitNormal (0.0f, 0.0f, 0.0f);
    RGB* firstHitAlbedoPtr = &firstHitAlbedo;
    Normal3D* firstHitNormalPtr = &firstHitNormal;

    // Initialize variables for storing hit point properties
    HitPointParams hitPointParams;
    hitPointParams.positionInWorld = Point3D (NAN);
    hitPointParams.prevPositionInWorld = Point3D (NAN);
    hitPointParams.normalInWorld = Normal3D (NAN);
    hitPointParams.texCoord = Point2D (NAN);
    hitPointParams.materialSlot = 0xFFFFFFFF;
    HitPointParams* hitPointParamsPtr = &hitPointParams;

    // Main path tracing loop
    while (true)
    {
        // Trace the ray and collect results
        SearchRayPayloadSignature::trace (
            plp.travHandle, origin.toNative(), direction.toNative(),
            0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
            RayType_Search, NumRayTypes, RayType_Search,
            rng, payloadPtr, hitPointParamsPtr, firstHitAlbedoPtr, firstHitNormalPtr);

        // Break out of the loop if conditions are met
        if (payload.terminate || payload.pathLength >= 10)
            break;

        // Update ray origin and direction for the next iteration
        origin = payload.origin;
        direction = payload.direction;
        ++payload.pathLength;
    }

    // Store the updated RNG state back to the buffer
    plp.rngBuffer[launchIndex] = rng;

    // Output information for the denoiser
    firstHitNormal = transpose (camera.orientation) * hitPointParams.normalInWorld;
    firstHitNormal.x *= -1;

    RGB prevAlbedoResult (0.0f, 0.0f, 0.0f);
    RGB prevColorResult (0.0f, 0.0f, 0.0f);
    Normal3D prevNormalResult (0.0f, 0.0f, 0.0f);

    if (plp.numAccumFrames > 0)
    {
        prevColorResult = RGB (getXYZ (plp.colorAccumBuffer.read (launchIndex)));
        prevAlbedoResult = RGB (getXYZ (plp.albedoAccumBuffer.read (launchIndex)));
        prevNormalResult = Normal3D (getXYZ (plp.normalAccumBuffer.read (launchIndex)));
    }

    float curWeight = 1.0f / (1 + plp.numAccumFrames);
    RGB colorResult = (1 - curWeight) * prevColorResult + curWeight * payload.contribution;
#if 1
    if (isnan (colorResult.r) || isnan (colorResult.g) || isnan (colorResult.b))
    {
        // Add this line to print the payload.contribution values
        printf ("payload.contribution: %f, %f, %f\n", payload.contribution.r, payload.contribution.g, payload.contribution.b);
        colorResult = RGB (make_float3 (1000000.0f, 0.0f, 0.0f)); // super red
    }
    else if (isinf (colorResult.r) || isinf (colorResult.g) || isinf (colorResult.b))
    {
        printf ("payload.contribution: %f, %f, %f\n", payload.contribution.r, payload.contribution.g, payload.contribution.b);
        colorResult = RGB (make_float3 (0.0f, 1000000.0f, 0.0f)); // super green
    }
    else if (colorResult.r < 0.0f || colorResult.g < 0.0f || colorResult.b < 0.0f)
    {
        printf ("payload.contribution: %f, %f, %f\n", payload.contribution.r, payload.contribution.g, payload.contribution.b);
        colorResult = RGB (make_float3 (0.0f, 0.0f, 1000000.0f)); // super blue
    }
#endif
    RGB albedoResult = (1 - curWeight) * prevAlbedoResult + curWeight * firstHitAlbedo;
    Normal3D normalResult = (1 - curWeight) * prevNormalResult + curWeight * firstHitNormal;

    plp.colorAccumBuffer.write (launchIndex, make_float4 (colorResult.toNative(), 1.0f));
    plp.albedoAccumBuffer.write (launchIndex, make_float4 (albedoResult.toNative(), 1.0f));
    plp.normalAccumBuffer.write (launchIndex, make_float4 (normalResult.toNative(), 1.0f));
}

// CUDA kernel for a miss progrm in OptiX
CUDA_DEVICE_KERNEL void RT_MS_NAME (miss)()
{
    // Declare pointers for the ray payload and hit point parameters
    SearchRayPayload* payload;
    HitPointParams* hitPntParams;

    // Retrieve ray payload and hit point parameters
    SearchRayPayloadSignature::get (nullptr, &payload, &hitPntParams, nullptr, nullptr);

    // If there's no environment light texture, apply some basic ambient light and terminate
    if (plp.envLightTexture == 0)
    {
        payload->contribution += payload->alpha * RGB (0.01f, 0.015f, 0.02f);
        payload->terminate = true;
        return;
    }
    // Store the normalized direction as the surface normal
    // Get the inverse of the ray direction in world space
    Vector3D vOut (-Vector3D (optixGetWorldRayDirection()));
    hitPntParams->normalInWorld = Normal3D (vOut);

    Vector3D rayDir = normalize (Vector3D (optixGetWorldRayDirection()));
    float posPhi, theta;
    toPolarYUp (rayDir, &posPhi, &theta);

    float phi = posPhi + plp.envLightRotation;
    phi = phi - floorf (phi / (2 * Pi)) * 2 * Pi;
    Point2D texCoord (phi / (2 * Pi), theta / Pi);

    float4 texValue = tex2DLod<float4> (plp.envLightTexture, texCoord.x, texCoord.y, 0.0f);
    RGB luminance = plp.envLightPowerCoeff * RGB (texValue.x, texValue.y, texValue.z);
    float misWeight = 1.0f;
    if (payload->pathLength > 1) // coming off a surface
    {
        float uvPDF = plp.envLightImportanceMap.evaluatePDF (texCoord.x, texCoord.y);
        float hypAreaPDensity = uvPDF / (2 * Pi * Pi * std::sin (theta));
        // FIXME

        float lightPDensity =
            (plp.lightInstDist.integral() > 0.0f ? probToSampleEnvLight : 1.0f) *
            hypAreaPDensity;
        // FIXME
        // float bsdfPDensity = rwPayload->prevDirPDensity;
        float bsdfPDensity = 0.25f; // just guessing
        misWeight = pow2 (bsdfPDensity) / (pow2 (bsdfPDensity) + pow2 (lightPDensity));

        payload->contribution += payload->alpha * luminance * misWeight;
    }
    else
        payload->contribution = luminance;

    // Terminate the ray
    payload->terminate = true;
}

CUDA_DEVICE_KERNEL void RT_CH_NAME (shading)()
{
    // Retrieve material and geometry data from SBT (Shader Binding Table)
    auto sbtr = HitGroupSBTRecordData::get();
    const Shared::MaterialData& mat = sbtr.matData;
    const GeometryData& geom = sbtr.geomData;

    // Initialize random number generator and payload
    PCG32RNG rng;
    SearchRayPayload* payload;
    RGB* firstHitAlbedo;
    Normal3D* firstHitNormal;
    HitPointParams* hitPntParams;
    SearchRayPayloadSignature::get (&rng, &payload, &hitPntParams, &firstHitAlbedo, &firstHitNormal);

    // Retrieve ray origin in world coordinates
    const Point3D rayOrigin (optixGetWorldRayOrigin());

    // Get hit point parameters and compute various surface attributes
    auto hp = HitPointParameter::get();
    Point3D positionInWorld;
    Normal3D shadingNormalInWorld;
    Vector3D texCoord0DirInWorld;
    Normal3D geometricNormalInWorld;
    Point2D texCoord;
    float hypAreaPDensity;
    computeSurfacePoint (
        geom, hp.primIndex, hp.b1, hp.b2,
        rayOrigin,
        &positionInWorld, &shadingNormalInWorld, &texCoord0DirInWorld,
        &geometricNormalInWorld, &texCoord, &hypAreaPDensity);

    // Compute outgoing direction in world coordinates and check hit side
    Vector3D vOut = normalize (-Vector3D (optixGetWorldRayDirection()));
    float frontHit = dot (vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;

    // Create a reference frame for shading based on shading normal
    ReferenceFrame shadingFrame (shadingNormalInWorld, texCoord0DirInWorld);

    // Offset the hit point along the normal to avoid self-intersection
    positionInWorld = offsetRayOrigin (positionInWorld, frontHit * geometricNormalInWorld);
    Vector3D vOutLocal = shadingFrame.toLocal (vOut);

    // Fetch or calculate albedo
    RGB albedo;
    if (mat.texture)
        albedo = RGB (getXYZ (tex2DLod<float4> (mat.texture, texCoord.x, texCoord.y, 0.0f)));
    else
        albedo = RGB (mat.albedo);

    // Create Lambertian BRDF
    LambertBRDF bsdf (albedo);

    // Perform Next Event Estimation for direct lighting
    payload->contribution += payload->alpha * performNextEventEstimation (
                                                  positionInWorld, vOutLocal, shadingFrame, bsdf, rng);

    // Generate the next ray for path tracing
    Vector3D vInLocal;
    float dirPDensity;
    payload->alpha *= bsdf.sampleThroughput (
        vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &vInLocal, &dirPDensity);
    Vector3D vIn = shadingFrame.fromLocal (vInLocal);

    // Store hit point data for subsequent usage
    hitPntParams->normalInWorld = shadingNormalInWorld;
    if (payload->pathLength == 1)
    {
        *firstHitAlbedo = albedo;
        *firstHitNormal = shadingNormalInWorld;
    }

    // Update payload with new ray data
    payload->origin = positionInWorld;
    payload->direction = vIn;
    payload->terminate = false;

    // Finalize payload
    SearchRayPayloadSignature::set (&rng, nullptr, nullptr, nullptr, nullptr);
}

CUDA_DEVICE_KERNEL void RT_AH_NAME (visibility)()
{
    float visibility = 0.0f;
    VisibilityRayPayloadSignature::set (&visibility);

    optixTerminateRay();
}
