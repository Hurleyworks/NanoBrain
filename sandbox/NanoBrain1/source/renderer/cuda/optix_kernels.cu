#include "hip/hip_runtime.h"
﻿#pragma once

// taken from OptiX_Utility
// https://github.com/shocker-0x15/OptiX_Utility/blob/master/LICENSE.md

#include "../Shared.h"

using namespace Shared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters plp;

struct HitPointParameter
{
    float b1, b2;
    int32_t primIndex;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static HitPointParameter get()
    {
        HitPointParameter ret;
        float2 bc = optixGetTriangleBarycentrics();
        ret.b1 = bc.x;
        ret.b2 = bc.y;
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

struct HitGroupSBTRecordData
{
    GeometryData geomData;
    MaterialData matData;

    CUDA_DEVICE_FUNCTION CUDA_INLINE static const HitGroupSBTRecordData& get()
    {
        return *reinterpret_cast<HitGroupSBTRecordData*> (optixGetSbtDataPointer());
    }
};

CUDA_DEVICE_KERNEL void RT_RG_NAME (pathTracing)()
{
    uint2 launchIndex = make_uint2 (optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    PCG32RNG rng = plp.rngBuffer[launchIndex];

    float jx = rng.getFloat0cTo1o();
    float jy = rng.getFloat0cTo1o();
    plp.rngBuffer.write (launchIndex, rng);

    float x = (launchIndex.x + jx) / plp.imageSize.x;
    float y = (launchIndex.y + jy) / plp.imageSize.y;
    float vh = 2 * std::tan (plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    float3 origin = plp.camera.position;
    float3 direction = normalize (plp.camera.orientation * make_float3 (vw * (0.5f - x), vh * (0.5f - y), 1));

    SearchRayPayload payload;
    payload.alpha = make_float3 (1.0f, 1.0f, 1.0f);
    payload.contribution = make_float3 (0.0f, 0.0f, 0.0f);
    payload.pathLength = 1;
    payload.terminate = false;
    SearchRayPayload* payloadPtr = &payload;
    float3 firstHitAlbedo = make_float3 (0.0f, 0.0f, 0.0f);
    float3 firstHitNormal = make_float3 (0.0f, 0.0f, 0.0f);
    float3* firstHitAlbedoPtr = &firstHitAlbedo;
    float3* firstHitNormalPtr = &firstHitNormal;
    while (true)
    {
        SearchRayPayloadSignature::trace (
            plp.travHandle, origin, direction,
            0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
            RayType_Search, NumRayTypes, RayType_Search,
            rng, payloadPtr, firstHitAlbedoPtr, firstHitNormalPtr);
        if (payload.terminate || payload.pathLength >= 10)
            break;

        origin = payload.origin;
        direction = payload.direction;
        ++payload.pathLength;
    }

    plp.rngBuffer[launchIndex] = rng;

    if (plp.useCameraSpaceNormal)
    {
        // Convert the normal into the camera space (right handed, looking down the negative Z-axis).
        firstHitNormal = transpose (plp.camera.orientation) * firstHitNormal;
        firstHitNormal.x *= -1;
    }

    float3 prevColorResult = make_float3 (0.0f, 0.0f, 0.0f);
    float3 prevAlbedoResult = make_float3 (0.0f, 0.0f, 0.0f);
    float3 prevNormalResult = make_float3 (0.0f, 0.0f, 0.0f);
    if (plp.numAccumFrames > 0)
    {
        prevColorResult = getXYZ (plp.colorAccumBuffer.read (launchIndex));
        prevAlbedoResult = getXYZ (plp.albedoAccumBuffer.read (launchIndex));
        prevNormalResult = getXYZ (plp.normalAccumBuffer.read (launchIndex));
    }
    float curWeight = 1.0f / (1 + plp.numAccumFrames);
    float3 colorResult = (1 - curWeight) * prevColorResult + curWeight * payload.contribution;
    float3 albedoResult = (1 - curWeight) * prevAlbedoResult + curWeight * firstHitAlbedo;
    float3 normalResult = (1 - curWeight) * prevNormalResult + curWeight * firstHitNormal;
    plp.colorAccumBuffer.write (launchIndex, make_float4 (colorResult, 1.0f));
    plp.albedoAccumBuffer.write (launchIndex, make_float4 (albedoResult, 1.0f));
    plp.normalAccumBuffer.write (launchIndex, make_float4 (normalResult, 1.0f));
}

CUDA_DEVICE_FUNCTION void toPolarYUp (const float3& v, float* phi, float* theta)
{
    *theta = std::acos (min (max (v.y, -1.0f), 1.0f));
    *phi = std::fmod (std::atan2 (-v.x, v.z) + 2 * Pi,
                      2 * Pi);
}

CUDA_DEVICE_KERNEL void RT_MS_NAME (miss)()
{
    SearchRayPayload* payload;
    float3* albedo;
    float3* normal;
    SearchRayPayloadSignature::get (nullptr, &payload, &albedo, &normal);

    if (plp.envLightTexture == 0)
    {
        payload->contribution += payload->alpha * make_float3 (0.01f, 0.015f, 0.02f);
        payload->terminate = true;
        return;
    }

    float3 p = optixGetWorldRayDirection();

    float posPhi, posTheta;
    toPolarYUp (p, &posPhi, &posTheta);

    float phi = posPhi + plp.envLightRotation;

    float u = phi / (2 * Pi);
    u -= floorf (u);
    float v = posTheta / Pi;

    //  if (plp.numAccumFrames == 2)
    //     pixelprintf (launchIndex, 100, 100, "%f-%f-%u \n", u, v, plp.numAccumFrames);

    float4 texValue = tex2DLod<float4> (plp.envLightTexture, u, v, 0.0f);
    float3 luminance = make_float3 (texValue);
    luminance *= plp.envLightPowerCoeff;

    payload->contribution += payload->alpha * luminance / Pi;

    payload->terminate = true;
}

CUDA_DEVICE_KERNEL void RT_CH_NAME (shading)()
{
    auto sbtr = HitGroupSBTRecordData::get();
    const MaterialData& mat = sbtr.matData;
    const GeometryData& geom = sbtr.geomData;

    PCG32RNG rng;
    SearchRayPayload* payload;
    float3* firstHitAlbedo;
    float3* firstHitNormal;
    SearchRayPayloadSignature::get (&rng, &payload, &firstHitAlbedo, &firstHitNormal);

    auto hp = HitPointParameter::get();
    float3 hitPointWorld;
    float3 surfaceNormalWorld;
    float2 texCoord;
    {
        const Triangle& tri = geom.triangleBuffer[hp.primIndex];
        const Vertex& v0 = geom.vertexBuffer[tri.index0];
        const Vertex& v1 = geom.vertexBuffer[tri.index1];
        const Vertex& v2 = geom.vertexBuffer[tri.index2];
        float b1 = hp.b1;
        float b2 = hp.b2;
        float b0 = 1 - (b1 + b2);
        hitPointWorld = b0 * v0.position + b1 * v1.position + b2 * v2.position;
        surfaceNormalWorld = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
        texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

        hitPointWorld = optixTransformPointFromObjectToWorldSpace (hitPointWorld);
        surfaceNormalWorld = normalize (optixTransformNormalFromObjectToWorldSpace (surfaceNormalWorld));
    }

    // From ChatGPT4
    // Basically, this chunk of code helps determine if the ray hits the front or back
    // of a surface and adjusts the normal and hit point accordingly.

    // Ah, got it. If you used the incoming ray direction optixGetWorldRayDirection()
    // instead of its negative -optixGetWorldRayDirection() for isFrontFace, the logic would flip.
    // Specifically, a positive dot product would then indicate a back-face hit, and a negative
    // or zero would indicate a front-face hit. This is because the incoming ray direction and
    // the surface normal would be pointing in roughly the same direction for back-face hits,
    // making the dot product positive.

    // So yes, you could use the incoming ray direction, but you'd need to
    // adjust the logic of the isFrontFace check accordingly.

    // vOut is the opposite of the incoming ray direction in world coordinates.
    float3 vOut = -optixGetWorldRayDirection();

    // This line checks if the ray hits the front face of the surface.
    // If the dot product is positive, it means the surface normal and
    // the ray direction are somewhat aligned, indicating a front-face hit.
    bool isFrontFace = dot (vOut, surfaceNormalWorld) > 0;

    // If it's not a front-face hit, the code flips the normal to point outward.
    if (!isFrontFace)
        surfaceNormalWorld = -surfaceNormalWorld;

    // This nudges the hit point slightly along the surface normal to
    // avoid self-intersection in future ray casts.
    hitPointWorld = hitPointWorld + surfaceNormalWorld * 0.001f;

    float3 albedo;
    if (mat.texture)
        albedo = getXYZ (tex2DLod<float4> (mat.texture, texCoord.x, texCoord.y, 0.0f));
    else
        albedo = mat.albedo;

    if (payload->pathLength == 1)
    {
        *firstHitAlbedo = albedo;
        *firstHitNormal = surfaceNormalWorld;
    }
   
    // From ChatGPT4
    // Here's a simplified example of how you might sample the environment map
    // based on the surface normal to simulate Lambertian reflection:
    float posPhi, posTheta;
    toPolarYUp (surfaceNormalWorld, &posPhi, &posTheta);

    float ph = posPhi + plp.envLightRotation;

    float u = ph / (2 * Pi);
    u -= floorf (u);
    float v = posTheta / Pi;

    float4 texValue = tex2DLod<float4> (plp.envLightTexture, u, v, 0.0f);
    float3 environmentLight = make_float3 (texValue);
    environmentLight *= plp.envLightPowerCoeff;

    float3 lambertReflection = environmentLight / Pi;

    // Update payload's contribution using Lambert's reflection
    payload->contribution += payload->alpha * albedo * lambertReflection;

    // This lambda function generates a local coordinate system (s, t, n) based on the given normal n.
    // This is useful for transforming vectors from one coordinate system to another, like from
    // the global coordinate system to a surface-local one.
    const auto makeCoordinateSystem = [] (const float3& n, float3* s, float3* t)
    {
        // Here, sign, a, and b are calculated to construct the s and t vectors.
        // They are part of the mathematical magic that simplifies the coordinate system generation..
        float sign = n.z >= 0 ? 1 : -1;
        float a = -1 / (sign + n.z);
        float b = n.x * n.y * a;

        // s and t are the local coordinate system basis vectors perpendicular to n.
        *s = make_float3 (1 + sign * n.x * n.x * a, sign * b, -sign * n.x);
        *t = make_float3 (b, sign + n.y * n.y * a, -n.y);
    };

    float3 s;
    float3 t;
    makeCoordinateSystem (surfaceNormalWorld, &s, &t);

    // generate random incoming direction
    // phi and theta are random angles, generated to produce a random incoming direction vIn.
    float phi = 2 * Pi * rng.getFloat0cTo1o();
    float theta = std::asin (std::sqrt (rng.getFloat0cTo1o()));
    float sinTheta = std::sin (theta);

    // Here, vIn is calculated in spherical coordinates. This is a general way to express directions in 3D space.
    float3 vIn = make_float3 (std::cos (phi) * sinTheta, std::sin (phi) * sinTheta, std::cos (theta));

    // the ultimate goal is to get vIn in world coordinates, but this code is doing it in a roundabout way.
    // It initially calculates vIn in a local coordinate system where the surface normal (surfaceNormalWorld) is the z-axis.
    // This makes the math for random sampling easier.

    // This code is actually converting vIn back to world coordinates.
    // It does this by taking the vIn defined in this local coordinate system
    // and dotting it with each of the basis vectors (s, t, surfaceNormalWorld) to get the
    // components in the world coordinates.

    // So, to sum it up, vIn starts out in a convenient local coordinate system for the calculations
    // and then gets transformed back to world coordinates.
    vIn = make_float3 (dot (make_float3 (s.x, t.x, surfaceNormalWorld.x), vIn),
                       dot (make_float3 (s.y, t.y, surfaceNormalWorld.y), vIn),
                       dot (make_float3 (s.z, t.z, surfaceNormalWorld.z), vIn));

    payload->alpha = payload->alpha * albedo;
    payload->origin = hitPointWorld;
    payload->direction = vIn;
    payload->terminate = false;

    SearchRayPayloadSignature::set (&rng, nullptr, nullptr, nullptr);
}

CUDA_DEVICE_KERNEL void RT_AH_NAME (visibility)()
{
    float visibility = 0.0f;
    VisibilityRayPayloadSignature::set (&visibility);

    optixTerminateRay();
}
