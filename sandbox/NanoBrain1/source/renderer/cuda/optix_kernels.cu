#include "hip/hip_runtime.h"
﻿#pragma once

// taken from OptiX_Utility
// https://github.com/shocker-0x15/OptiX_Utility/blob/master/LICENSE.md
// and from Shocker GfxExp
// https://github.com/shocker-0x15/GfxExp

#include "../Shared.h"
#include <common_device.h>

using namespace Shared;
using namespace shared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters plp;

static constexpr bool useSolidAngleSampling = false;
static constexpr bool useImplicitLightSampling = true;
static constexpr bool useExplicitLightSampling = true;
static constexpr bool useMultipleImportanceSampling = useImplicitLightSampling && useExplicitLightSampling;
static_assert (useImplicitLightSampling || useExplicitLightSampling, "Invalid configuration for light sampling.");

class LambertBRDF
{
    RGB m_reflectance;

 public:
    CUDA_DEVICE_FUNCTION LambertBRDF (const RGB& reflectance) :
        m_reflectance (reflectance) {}

    CUDA_DEVICE_FUNCTION void getSurfaceParameters (RGB* diffuseReflectance, RGB* specularReflectance, float* roughness) const
    {
        *diffuseReflectance = m_reflectance;
        *specularReflectance = RGB (0.0f, 0.0f, 0.0f);
        *roughness = 1.0f;
    }

    CUDA_DEVICE_FUNCTION RGB sampleThroughput (
        const Vector3D& vGiven, float uDir0, float uDir1,
        Vector3D* vSampled, float* dirPDensity) const
    {
        *vSampled = cosineSampleHemisphere (uDir0, uDir1);
        *dirPDensity = vSampled->z / Pi;
        if (vGiven.z <= 0.0f)
            vSampled->z *= -1;
        return m_reflectance;
    }

    CUDA_DEVICE_FUNCTION RGB evaluate (const Vector3D& vGiven, const Vector3D& vSampled) const
    {
        if (vGiven.z * vSampled.z > 0)
            return m_reflectance / Pi;
        else
            return RGB (0.0f, 0.0f, 0.0f);
    }
    CUDA_DEVICE_FUNCTION float evaluatePDF (const Vector3D& vGiven, const Vector3D& vSampled) const
    {
        if (vGiven.z * vSampled.z > 0)
            return fabs (vSampled.z) / Pi;
        else
            return 0.0f;
    }

    CUDA_DEVICE_FUNCTION RGB evaluateDHReflectanceEstimate (const Vector3D& vGiven) const
    {
        return m_reflectance;
    }
};


template <bool computeHypotheticalAreaPDensity, bool useSolidAngleSampling>
CUDA_DEVICE_FUNCTION CUDA_INLINE void computeSurfacePoint (
    const Shared::GeometryData& geomInst,
    uint32_t primIndex, float b1, float b2,
    const Point3D& referencePoint,
    Point3D* positionInWorld, Normal3D* shadingNormalInWorld, Vector3D* texCoord0DirInWorld,
    Normal3D* geometricNormalInWorld, Point2D* texCoord,
    float* hypAreaPDensity)
{
    const Triangle& tri = geomInst.triangleBuffer[primIndex];
    const Vertex& v0 = geomInst.vertexBuffer[tri.index0];
    const Vertex& v1 = geomInst.vertexBuffer[tri.index1];
    const Vertex& v2 = geomInst.vertexBuffer[tri.index2];
    const Point3D p[3] = {
        transformPointFromObjectToWorldSpace (v0.position),
        transformPointFromObjectToWorldSpace (v1.position),
        transformPointFromObjectToWorldSpace (v2.position),
    };
    float b0 = 1 - (b1 + b2);

    // EN: Compute hit point properties in the local coordinates.
    *positionInWorld = b0 * p[0] + b1 * p[1] + b2 * p[2];
    Normal3D shadingNormal = b0 * v0.normal + b1 * v1.normal + b2 * v2.normal;
    Vector3D texCoord0Dir = b0 * v0.texCoord0Dir + b1 * v1.texCoord0Dir + b2 * v2.texCoord0Dir;
    Normal3D geometricNormal (cross (p[1] - p[0], p[2] - p[0]));
    float area;
    if constexpr (computeHypotheticalAreaPDensity && !useSolidAngleSampling)
        area = 0.5f * length (geometricNormal);
    else
        (void)area;
    *texCoord = b0 * v0.texCoord + b1 * v1.texCoord + b2 * v2.texCoord;

    // EN: Convert the local properties to ones in world coordinates.
    *shadingNormalInWorld = normalize (transformNormalFromObjectToWorldSpace (shadingNormal));
    *texCoord0DirInWorld = normalize (transformVectorFromObjectToWorldSpace (texCoord0Dir));
    *geometricNormalInWorld = normalize (geometricNormal);
    if (!shadingNormalInWorld->allFinite())
    {
        *shadingNormalInWorld = Normal3D (0, 0, 1);
        *texCoord0DirInWorld = Vector3D (1, 0, 0);
    }
    if (!texCoord0DirInWorld->allFinite())
    {
        Vector3D bitangent;
        makeCoordinateSystem (*shadingNormalInWorld, texCoord0DirInWorld, &bitangent);
    }

    if constexpr (computeHypotheticalAreaPDensity)
    {
        // EN: Compute a hypothetical probability density with which the intersection point
        //     is sampled by explicit light sampling.
        float lightProb = 1.0f;
        if (plp.envLightTexture && plp.enableEnvLight)
            lightProb *= (1 - probToSampleEnvLight);

        if (!isfinite (lightProb))
        {
            *hypAreaPDensity = 0.0f;
            return;
        }

        *hypAreaPDensity = lightProb / area;
    }
}

// Define a struct called HitPointParameter to hold hit point info
struct HitPointParameter
{
    float b1, b2;      // Barycentric coordinates
    int32_t primIndex; // Index of the primitive hit by the ray

    // Static member function to get hit point parameters
    CUDA_DEVICE_FUNCTION CUDA_INLINE static HitPointParameter get()
    {
        HitPointParameter ret; // Create an instance of the struct

        // Get barycentric coordinates from OptiX API
        float2 bc = optixGetTriangleBarycentrics();

        // Store the barycentric coordinates in the struct
        ret.b1 = bc.x;
        ret.b2 = bc.y;

        // Get the index of the primitive hit by the ray from OptiX API
        ret.primIndex = optixGetPrimitiveIndex();

        // Return the populated struct
        return ret;
    }
};
;

// This struct is used to fetch geometry and material data from
// the Shader Binding Table (SBT) in OptiX.
struct HitGroupSBTRecordData
{
    GeometryData geomData;        // Geometry data for the hit object
    Shared::MaterialData matData; // Material data for the hit object

    // Static member function to retrieve the SBT record data
    CUDA_DEVICE_FUNCTION CUDA_INLINE static const HitGroupSBTRecordData& get()
    {
        // Use optixGetSbtDataPointer() to get the pointer to the SBT data
        // Cast the pointer to type HitGroupSBTRecordData and dereference it
        return *reinterpret_cast<HitGroupSBTRecordData*> (optixGetSbtDataPointer());
    }
};

// Define the main CUDA device kernel for path tracing
CUDA_DEVICE_KERNEL void RT_RG_NAME (pathTracing)()
{
    // Get the launch index for this thread
    uint2 launchIndex = make_uint2 (optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    // Initialize the random number generator
    PCG32RNG rng = plp.rngBuffer[launchIndex];

    // Get camera properties from the pipeline parameters
    const PerspectiveCamera& camera = plp.camera;

    // Generate random numbers for jittering
    float jx = rng.getFloat0cTo1o();
    float jy = rng.getFloat0cTo1o();

    // Update the RNG buffer
    plp.rngBuffer.write (launchIndex, rng);

    // Compute normalized screen coordinates
    float x = (launchIndex.x + jx) / plp.imageSize.x;
    float y = (launchIndex.y + jy) / plp.imageSize.y;

    // Compute vertical and horizontal view angles
    float vh = 2 * std::tan (plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    // Setup ray origin and direction
    Point3D origin = camera.position;
    Vector3D direction = normalize (camera.orientation * Vector3D (vw * (0.5f - x), vh * (0.5f - y), 1));

    // Initialize ray payload
    SearchRayPayload payload;
    payload.alpha = RGB (1.0f, 1.0f, 1.0f);
    payload.contribution = RGB (0.0f, 0.0f, 0.0f);
    payload.pathLength = 1;
    payload.terminate = false;
    SearchRayPayload* payloadPtr = &payload;

    RGB firstHitAlbedo (0.0f, 0.0f, 0.0f);
    Normal3D firstHitNormal (0.0f, 0.0f, 0.0f);
    RGB* firstHitAlbedoPtr = &firstHitAlbedo;
    Normal3D* firstHitNormalPtr = &firstHitNormal;

    // Initialize variables for storing hit point properties
    HitPointParams hitPointParams;
    hitPointParams.positionInWorld = Point3D (NAN);
    hitPointParams.prevPositionInWorld = Point3D (NAN);
    hitPointParams.normalInWorld = Normal3D (NAN);
    hitPointParams.texCoord = Point2D (NAN);
    hitPointParams.materialSlot = 0xFFFFFFFF;
    HitPointParams* hitPointParamsPtr = &hitPointParams;

    // Main path tracing loop
    while (true)
    {
        // Trace the ray and collect results
        SearchRayPayloadSignature::trace (
            plp.travHandle, origin.toNative(), direction.toNative(),
            0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
            RayType_Search, NumRayTypes, RayType_Search,
            rng, payloadPtr, hitPointParamsPtr, firstHitAlbedoPtr, firstHitNormalPtr);

        // Break out of the loop if conditions are met
        if (payload.terminate || payload.pathLength >= 10)
            break;

        // Update ray origin and direction for the next iteration
        origin = payload.origin;
        direction = payload.direction;
        ++payload.pathLength;
    }

    // Store the updated RNG state back to the buffer
    plp.rngBuffer[launchIndex] = rng;

    // Output information for the denoiser
    firstHitNormal = transpose (camera.orientation) * hitPointParams.normalInWorld;
    firstHitNormal.x *= -1;

    RGB prevAlbedoResult (0.0f, 0.0f, 0.0f);
    RGB prevColorResult (0.0f, 0.0f, 0.0f);
    Normal3D prevNormalResult (0.0f, 0.0f, 0.0f);

    if (plp.numAccumFrames > 0)
    {
        prevColorResult = RGB (getXYZ (plp.colorAccumBuffer.read (launchIndex)));
        prevAlbedoResult = RGB (getXYZ (plp.albedoAccumBuffer.read (launchIndex)));
        prevNormalResult = Normal3D (getXYZ (plp.normalAccumBuffer.read (launchIndex)));
    }

    float curWeight = 1.0f / (1 + plp.numAccumFrames);
    RGB colorResult = (1 - curWeight) * prevColorResult + curWeight * payload.contribution;
    RGB albedoResult = (1 - curWeight) * prevAlbedoResult + curWeight * firstHitAlbedo;
    Normal3D normalResult = (1 - curWeight) * prevNormalResult + curWeight * firstHitNormal;

    plp.colorAccumBuffer.write (launchIndex, make_float4 (colorResult.toNative(), 1.0f));
    plp.albedoAccumBuffer.write (launchIndex, make_float4 (albedoResult.toNative(), 1.0f));
    plp.normalAccumBuffer.write (launchIndex, make_float4 (normalResult.toNative(), 1.0f));
}

// CUDA kernel for a miss progrm in OptiX
CUDA_DEVICE_KERNEL void RT_MS_NAME (miss)()
{
    // Declare pointers for the ray payload and hit point parameters
    SearchRayPayload* payload;
    HitPointParams* hitPntParams;

    // Retrieve ray payload and hit point parameters
    SearchRayPayloadSignature::get (nullptr, &payload, &hitPntParams, nullptr, nullptr);

    // If there's no environment light texture, apply some basic ambient light and terminate
    if (plp.envLightTexture == 0)
    {
        payload->contribution += payload->alpha * RGB (0.01f, 0.015f, 0.02f);
        payload->terminate = true;
        return;
    }
    // Store the normalized direction as the surface normal
    // Get the inverse of the ray direction in world space
    Vector3D vOut (-Vector3D (optixGetWorldRayDirection()));
    hitPntParams->normalInWorld = Normal3D (vOut);

    Vector3D rayDir = normalize (Vector3D (optixGetWorldRayDirection()));
    float posPhi, theta;
    toPolarYUp (rayDir, &posPhi, &theta);

    float phi = posPhi + plp.envLightRotation;
    phi = phi - floorf (phi / (2 * Pi)) * 2 * Pi;
    Point2D texCoord (phi / (2 * Pi), theta / Pi);

    
    float4 texValue = tex2DLod<float4> (plp.envLightTexture, texCoord.x, texCoord.y, 0.0f);
    RGB luminance = plp.envLightPowerCoeff * RGB (texValue.x, texValue.y, texValue.z);
    float misWeight = 1.0f;
    if constexpr (true)
    {
        float uvPDF = plp.envLightImportanceMap.evaluatePDF (texCoord.x, texCoord.y);
        float hypAreaPDensity = uvPDF / (2 * Pi * Pi * std::sin (theta));
        // FIXME

        float lightPDensity =
            (plp.lightInstDist.integral() > 0.0f ? probToSampleEnvLight : 1.0f) *
            hypAreaPDensity;
        // FIXME
        // float bsdfPDensity = rwPayload->prevDirPDensity;
        float bsdfPDensity = 0.5f; // just guessing
        misWeight = pow2 (bsdfPDensity) / (pow2 (bsdfPDensity) + pow2 (lightPDensity));
    }
    payload->contribution += payload->alpha * luminance * misWeight;

    // Terminate the ray
    payload->terminate = true;
}

// CUDA kernel for a closest hit progrm in OptiX
CUDA_DEVICE_KERNEL void RT_CH_NAME (shading)()
{
    auto sbtr = HitGroupSBTRecordData::get();
    const Shared::MaterialData& mat = sbtr.matData;
    const GeometryData& geom = sbtr.geomData;

    PCG32RNG rng;
    SearchRayPayload* payload;
    RGB* firstHitAlbedo;
    Normal3D* firstHitNormal;
    HitPointParams* hitPntParams;
    SearchRayPayloadSignature::get (&rng, &payload, &hitPntParams, &firstHitAlbedo, &firstHitNormal);

    const Point3D rayOrigin (optixGetWorldRayOrigin());

    auto hp = HitPointParameter::get();
    Point3D positionInWorld;
    Normal3D shadingNormalInWorld;
    Vector3D texCoord0DirInWorld;
    Normal3D geometricNormalInWorld;
    Point2D texCoord;
    float hypAreaPDensity;
    computeSurfacePoint<useMultipleImportanceSampling, useSolidAngleSampling> (
        geom, hp.primIndex, hp.b1, hp.b2,
        rayOrigin,
        &positionInWorld, &shadingNormalInWorld, &texCoord0DirInWorld,
        &geometricNormalInWorld, &texCoord, &hypAreaPDensity);
    if constexpr (!useMultipleImportanceSampling)
        (void)hypAreaPDensity;

    Vector3D vOut = normalize (-Vector3D (optixGetWorldRayDirection()));
    float frontHit = dot (vOut, geometricNormalInWorld) >= 0.0f ? 1.0f : -1.0f;

    ReferenceFrame shadingFrame (shadingNormalInWorld, texCoord0DirInWorld);

    positionInWorld = offsetRayOrigin (positionInWorld, frontHit * geometricNormalInWorld);
    Vector3D vOutLocal = shadingFrame.toLocal (vOut);

    RGB albedo;
    if (mat.texture)
        albedo = RGB (getXYZ (tex2DLod<float4> (mat.texture, texCoord.x, texCoord.y, 0.0f)));
    else
        albedo = RGB (mat.albedo);

    // generate next ray.
    Vector3D vInLocal;
    float dirPDensity;
    LambertBRDF bsdf (albedo);
    payload->alpha *= bsdf.sampleThroughput (
        vOutLocal, rng.getFloat0cTo1o(), rng.getFloat0cTo1o(),
        &vInLocal, &dirPDensity);
    Vector3D vIn = shadingFrame.fromLocal (vInLocal);

    // A simplified example of how you might sample the environment map
    // based on the surface normal to simulate Lambertian reflection:
    float posPhi, posTheta;
    toPolarYUp (Vector3D (shadingNormalInWorld), &posPhi, &posTheta);

    float ph = posPhi + plp.envLightRotation;

    float u = ph / (2 * Pi);
    u -= floorf (u);
    float v = posTheta / Pi;

    float4 texValue = tex2DLod<float4> (plp.envLightTexture, u, v, 0.0f);
    RGB environmentLight (texValue.x, texValue.y, texValue.z);
    environmentLight *= plp.envLightPowerCoeff;

    RGB lambertReflection (environmentLight / Pi);

    // Update payload's contribution using Lambert's reflection
    payload->contribution += payload->alpha * albedo * lambertReflection;

    hitPntParams->normalInWorld = shadingNormalInWorld;
    if (payload->pathLength == 1)
    {
        *firstHitAlbedo = albedo;
        *firstHitNormal = shadingNormalInWorld;
    }

    payload->origin = positionInWorld;
    payload->direction = vIn;
    payload->terminate = false;

    SearchRayPayloadSignature::set (&rng, nullptr, nullptr, nullptr, nullptr);
}

CUDA_DEVICE_KERNEL void RT_AH_NAME (visibility)()
{
    float visibility = 0.0f;
    VisibilityRayPayloadSignature::set (&visibility);

    optixTerminateRay();
}
